#include <stdio.h>
#include <stdlib.h>
#include <cassert>
#include <hip/hip_runtime.h>
#include <chrono>
#include <unistd.h>
#include <iostream>

const int INF = ((1 << 30) - 1); 
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B);
int ceil(int a, int b);
template <int,int> __global__ void phase1(int* Dist_gpu, int Round, int n, int pitch_int);
template <int,int> __global__ void phase2_1(int* Dist_gpu, int Round, int n, int pitch_int);
template <int,int> __global__ void phase2_2(int* Dist_gpu, int Round, int n, int pitch_int);
template <int,int> __global__ void phase3(int* Dist_gpu, int Round, int n, int pitch_int);

int n, m;   // Number of vertices, edges
int* Dist;
int* Dist_gpu;
size_t pitch;

int N;

int main(int argc, char* argv[])
{   
    
    input(argv[1]);
    int B = 64;
    block_FW(B);
    output(argv[2]);
    hipHostFree(Dist);
    hipFree(Dist_gpu);
    return 0;
}

void input(char* infile) { 
    FILE* file = fopen(infile, "rb"); 
    fread(&n, sizeof(int), 1, file); 
    fread(&m, sizeof(int), 1, file);

    N = n;
    n = (!n%64)? n : n + 64 - n%64;

    hipHostMalloc( &Dist, sizeof(int)*(n*n), hipHostMallocDefault);

    for (int i = 0; i < n; ++i) {
        int IN = i * n;
        #pragma GCC ivdep
        for (int j = 0; j < i; ++j) {
            Dist[IN + j] = INF;
        }
        #pragma GCC ivdep
        for (int j = i + 1; j < n; ++j) {
            Dist[IN + j] = INF;
        }
    }

    int pair[3]; 
    for (int i = 0; i < m; ++i) { 
        fread(pair, sizeof(int), 3, file); 
        Dist[pair[0] * n + pair[1]] = pair[2]; 
    } 
    fclose(file); 
}

void output(char *outFileName) {
    FILE *outfile = fopen(outFileName, "w");
    for (int i = 0; i < N; ++i) {
        fwrite(&Dist[i * n], sizeof(int), N, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) {
    return (a + b -1)/b;
}

void block_FW(int B)
{

    int round = ceil(n, B);
    hipMallocPitch((void**)&Dist_gpu, &pitch,n*sizeof(int), n+64);
    int pitch_int = pitch / sizeof(int);
    hipMemcpy2D(Dist_gpu, pitch, Dist, n*sizeof(int), n*sizeof(int), n, hipMemcpyHostToDevice);
    
    for (int r = 0; r < round; ++r) {
        
        switch(B){
            case 32:
            break;
            case 64:
                phase1  <64,4><<< 1                     , dim3(32,32),   64*64*sizeof(int) >>>(Dist_gpu, r, n, pitch_int);
                phase2_1<64,4><<< dim3(1, round-1)      , dim3(32,32), 2*64*64*sizeof(int) >>>(Dist_gpu, r, n, pitch_int);
                phase2_2<64,4><<< dim3(round-1, 1)      , dim3(32,32), 2*64*64*sizeof(int) >>>(Dist_gpu, r, n, pitch_int);
                phase3  <64,4><<< dim3(round-1, round-1), dim3(32,32), 2*64*64*sizeof(int) >>>(Dist_gpu, r, n, pitch_int);
            break;
        }
        
        
        
    }
    hipMemcpy2D(Dist, n*sizeof(int), Dist_gpu, pitch, n*sizeof(int), n, hipMemcpyDeviceToHost);
}

template <int B, int P>
__global__ 
void phase1(int* Dist_gpu, int Round, int n, int pitch_int) {

    extern __shared__ int shared_mem[]; 

    int sdx = (threadIdx.y * 64) + threadIdx.x;

    shared_mem[sdx]      = Dist_gpu[(Round * 64 + threadIdx.y)     *pitch_int + Round * 64 + threadIdx.x];
    shared_mem[sdx+32]   = Dist_gpu[(Round * 64 + threadIdx.y)     *pitch_int + Round * 64 + threadIdx.x + 32];
    shared_mem[sdx+2048] = Dist_gpu[(Round * 64 + threadIdx.y + 32)*pitch_int + Round * 64 + threadIdx.x];
    shared_mem[sdx+2080] = Dist_gpu[(Round * 64 + threadIdx.y + 32)*pitch_int + Round * 64 + threadIdx.x + 32];

    //__syncthreads();

    for(int k=0; k < 64; ++k){
        __syncthreads();
        shared_mem[sdx]      = min(shared_mem[sdx]     , shared_mem[threadIdx.y * 64 + k]    + shared_mem[k*64+threadIdx.x]);
        shared_mem[sdx+32]   = min(shared_mem[sdx+32]  , shared_mem[threadIdx.y * 64 + k]    + shared_mem[k*64+threadIdx.x + 32]);
        shared_mem[sdx+2048] = min(shared_mem[sdx+2048], shared_mem[(threadIdx.y+32)*64 + k] + shared_mem[k*64+threadIdx.x]);
        shared_mem[sdx+2080] = min(shared_mem[sdx+2080], shared_mem[(threadIdx.y+32)*64 + k] + shared_mem[k*64+threadIdx.x + 32]);
    }

    Dist_gpu[(Round * 64  + threadIdx.y)     *pitch_int + Round * 64 + threadIdx.x]       = shared_mem[sdx];
    Dist_gpu[(Round * 64  + threadIdx.y)     *pitch_int + Round * 64 + threadIdx.x + 32]  = shared_mem[sdx+32];
    Dist_gpu[(Round * 64  + threadIdx.y + 32)*pitch_int + Round * 64 + threadIdx.x]       = shared_mem[sdx+2048];
    Dist_gpu[(Round * 64  + threadIdx.y + 32)*pitch_int + Round * 64 + threadIdx.x + 32]  = shared_mem[sdx+2080];
}

template <int B, int P>
__global__ void phase2_1(int* Dist_gpu, int Round, int n, int pitch_int) {
    
    extern __shared__ int shared_mem[]; 

    int b_i = blockIdx.y + (blockIdx.y >= Round);
    int b_j = Round;

    int i = b_i * 64 + threadIdx.y;
    int j = b_j * 64 + threadIdx.x;

    int sdx = threadIdx.y * 64 + threadIdx.x;

    shared_mem[sdx]                    = Dist_gpu[i                             * pitch_int + j      ]; // IK
    shared_mem[sdx + 32]               = Dist_gpu[i                             * pitch_int + j + 32 ]; // IK
    shared_mem[sdx + 4096]             = Dist_gpu[(Round*64 + threadIdx.y)      * pitch_int + j      ]; // KJ
    shared_mem[sdx + 4128]             = Dist_gpu[(Round*64 + threadIdx.y)      * pitch_int + j + 32 ]; // KJ

    shared_mem[sdx + 2048]             = Dist_gpu[(i + 32)                      * pitch_int + j      ];
    shared_mem[sdx + 2080]             = Dist_gpu[(i + 32)                      * pitch_int + j + 32 ];
    shared_mem[sdx + 2048 + 4096]      = Dist_gpu[(Round*64 + threadIdx.y + 32) * pitch_int + j      ];
    shared_mem[sdx + 2080 + 4096]      = Dist_gpu[(Round*64 + threadIdx.y + 32) * pitch_int + j + 32 ];

    #pragma unroll
    for (int k = 0; k < 64; ++k) {
        __syncthreads();
        
        shared_mem[sdx]      = min(shared_mem[sdx],      shared_mem[threadIdx.y*B+k] + shared_mem[k*B+threadIdx.x + 4096]);
        shared_mem[sdx + 32] = min(shared_mem[sdx + 32], shared_mem[threadIdx.y*B+k] + shared_mem[k*B+threadIdx.x + 4128]);

        shared_mem[sdx + 2048] = min(shared_mem[sdx+2048], shared_mem[(threadIdx.y + 32)*64+k] + shared_mem[k*64+threadIdx.x + 4096]);
        shared_mem[sdx + 2080] = min(shared_mem[sdx+2080], shared_mem[(threadIdx.y + 32)*64+k] + shared_mem[k*64+threadIdx.x + 4128]);
    }

    Dist_gpu[i       *pitch_int + j     ] = shared_mem[sdx            ];  
    Dist_gpu[i       *pitch_int + j + 32] = shared_mem[sdx + 32       ];
    Dist_gpu[(i + 32)*pitch_int + j     ] = shared_mem[sdx + 2048     ];  
    Dist_gpu[(i + 32)*pitch_int + j + 32] = shared_mem[sdx + 2048 + 32]; 

}

template <int B, int P>
__global__ void phase2_2(int* Dist_gpu, int Round, int n, int pitch_int) {
    extern __shared__ int shared_mem[];

    int i = (Round << 6) + threadIdx.y;
    int j = ((blockIdx.x + (blockIdx.x >= Round)) << 6) + threadIdx.x;
    
    int sdx = threadIdx.y * 64 + threadIdx.x;

    shared_mem[sdx]                    = Dist_gpu[i*pitch_int + j];
    shared_mem[sdx + 32]               = Dist_gpu[i*pitch_int + j + 32];
    shared_mem[sdx + 4096]             = Dist_gpu[i*pitch_int + Round * 64 + threadIdx.x];
    shared_mem[sdx + 4096 + 32]        = Dist_gpu[i*pitch_int + Round * 64 + threadIdx.x + 32];

    shared_mem[sdx + 2048]             = Dist_gpu[(i + 32)*pitch_int + j];
    shared_mem[sdx + 2080]             = Dist_gpu[(i + 32)*pitch_int + j + 32];
    shared_mem[sdx + 2048 + 4096]      = Dist_gpu[(i + 32)*pitch_int + Round * 64 + threadIdx.x];
    shared_mem[sdx + 2080 + 4096]      = Dist_gpu[(i + 32)*pitch_int + Round * 64 + threadIdx.x + 32];

    

    #pragma unroll
    for (int k = 0; k < 64; ++k) {
        __syncthreads();

        
        shared_mem[sdx] = min(shared_mem[sdx], shared_mem[threadIdx.y*64+k+4096] + shared_mem[k*64+threadIdx.x]);
        shared_mem[sdx + 32] = min(shared_mem[sdx + 32], shared_mem[threadIdx.y*64+k+4096] + shared_mem[k*64+threadIdx.x + 32]);

        
        shared_mem[sdx+2048] = min(shared_mem[sdx+2048], shared_mem[(threadIdx.y+32)*64+k+4096] + shared_mem[k*64+threadIdx.x]);
        shared_mem[sdx+2080] = min(shared_mem[sdx+2080], shared_mem[(threadIdx.y+32)*64+k+4096] + shared_mem[k*64+threadIdx.x + 32]);
        
    }

    Dist_gpu[i*pitch_int + j]      = shared_mem[sdx];  
    Dist_gpu[i*pitch_int + j + 32] = shared_mem[sdx + 32];

    Dist_gpu[(i + 32)*pitch_int + j]      = shared_mem[sdx+2048];  
    Dist_gpu[(i + 32)*pitch_int + j + 32] = shared_mem[sdx+2048 + 32];
}

template <int B, int P>
__global__ void phase3(int* Dist_gpu, int Round, int n, int pitch_int) {

    int i = ((blockIdx.y + (blockIdx.y>=Round)) << 6) + threadIdx.y;
    int j = ((blockIdx.x + (blockIdx.x>=Round)) << 6) + threadIdx.x;

    extern __shared__ int shared_mem[];
    
    int d0 = Dist_gpu[i*pitch_int + j];
    int d1 = Dist_gpu[i*pitch_int + j + 32];
    int d2 = Dist_gpu[(i+32)*pitch_int + j];
    int d3 = Dist_gpu[(i+32)*pitch_int + j + 32];
    
    int sdx = threadIdx.y * 64 + threadIdx.x;

    shared_mem[ sdx ]       = Dist_gpu[i*pitch_int + Round * 64 + threadIdx.x];
    shared_mem[ sdx + 32]   = Dist_gpu[i*pitch_int + Round * 64 + threadIdx.x + 32];
    shared_mem[ sdx + 4096] = Dist_gpu[(Round * 64 + threadIdx.y)*pitch_int + j];
    shared_mem[ sdx + 4128] = Dist_gpu[(Round * 64 + threadIdx.y)*pitch_int + j + 32];

    sdx += 2048;

    shared_mem[ sdx ]       = Dist_gpu[(i + 32)*pitch_int + Round * 64 + threadIdx.x];
    shared_mem[ sdx + 32]   = Dist_gpu[(i + 32)*pitch_int + Round * 64 + threadIdx.x + 32];
    shared_mem[ sdx + 4096] = Dist_gpu[(Round * 64 + threadIdx.y + 32)*pitch_int + j];
    shared_mem[ sdx + 4128] = Dist_gpu[(Round * 64 + threadIdx.y + 32)*pitch_int + j + 32];

    __syncthreads();
    
    #pragma unroll
    for (int k = 0; k < 64; ++k) {
        int idx = threadIdx.y * 64 + k;
        int v0 = shared_mem[idx]        + shared_mem[k*64 + threadIdx.x + 4096];
        int v1 = shared_mem[idx]        + shared_mem[k*64 + threadIdx.x + 4128];
        int v2 = shared_mem[idx + 2048] + shared_mem[k*64 + threadIdx.x + 4096];
        int v3 = shared_mem[idx + 2048] + shared_mem[k*64 + threadIdx.x + 4128];
        d0 = min(d0, v0);
        d1 = min(d1, v1);
        d2 = min(d2, v2);
        d3 = min(d3, v3);
    }

    Dist_gpu[i*pitch_int + j]           = d0;
    Dist_gpu[i*pitch_int + j + 32]      = d1;
    Dist_gpu[(i+32)*pitch_int + j]      = d2;
    Dist_gpu[(i+32)*pitch_int + j + 32] = d3;

}

